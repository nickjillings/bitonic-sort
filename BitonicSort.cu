#include "hip/hip_runtime.h"

#include "BitonicSortCUDA.cuh"

// Bitonic for uint32_t
__device__ void _bitonicStep1_uint32(unsigned int * smem, int tid, int tpp, int d)
{
	int m = tid / (d >> 1);
	int tib = tid - m*(d >> 1);
	int addr1 = d*m + tib;
	int addr2 = (m + 1)*d - tib - 1;
	
	unsigned int A = smem[addr1];
	unsigned int B = smem[addr2];
	smem[addr1] = max(A, B);
	smem[addr2] = min(A, B);
}

__device__ void _bitonicStep2_uint32(unsigned int * smem, int tid, int tpp, int d)
{
	int m = tid / (d >> 1);
	int tib = tid - m*(d >> 1);
	int addr1 = d*m + tib;
	int addr2 = addr1 + (d >> 1);

	unsigned int A = smem[addr1];
	unsigned int B = smem[addr2];
	smem[addr1] = max(A, B);
	smem[addr2] = min(A, B);
}

__global__ void bitonicSortKernel128_uint32(unsigned int* mem)
{
	// Operating on 64 samples
	int bid = blockIdx.x; // Block UID
	int tpp = threadIdx.x; // Thread position in block
	int tid = blockIdx.x*blockDim.x + threadIdx.x; // Thread global UID
	__shared__ unsigned int smem[256]; // Two blocks worth of shared memory
	smem[tpp] = mem[blockDim.x*(2 * bid) + tpp]; // Coalesced memory load
	smem[tpp + blockDim.x] = mem[blockDim.x*((2 * bid) + 1) + tpp]; // Coalesced memory load
	int blocks = 8;
	for (int blockNum = 1; blockNum <= blocks; blockNum++)
	{
		int d = 1 << blockNum;
		_bitonicStep1_uint32(smem, tpp, tpp, d);
		__syncthreads();
		d = d >> 1;
		while(d >= 2)
		{
			_bitonicStep2_uint32(smem, tpp, tpp, d);
			__syncthreads();
			d = d >> 1;
		}
	}

	mem[blockDim.x*(2 * bid) + tpp] = smem[tpp];
	mem[blockDim.x*((2*bid)+1) + tpp] = smem[tpp + blockDim.x];
}

__global__ void bitonicSortKernelXBlock1_uint32(unsigned int* mem, int blockNum)
{
	int bid = blockIdx.x; // Block UID
	int tpp = threadIdx.x; // Thread position in block
	int tid = blockIdx.x*blockDim.x + threadIdx.x; // Thread global UID
	int d = 1 << blockNum;
	_bitonicStep1_uint32(mem, tid, tpp, d);
}
__global__ void bitonicSortKernelXBlock2_uint32(unsigned int* mem, int blockNum, int d)
{
	int bid = blockIdx.x; // Block UID
	int tpp = threadIdx.x; // Thread position in block
	int tid = blockIdx.x*blockDim.x + threadIdx.x; // Thread global UID
	_bitonicStep2_uint32(mem, tid, tpp, d);
	
}


// For float32

__device__ void _bitonicStep1_fp32(float * smem, int tid, int tpp, int d)
{
	int m = tid / (d >> 1);
	int tib = tid - m*(d >> 1);
	int addr1 = d*m + tib;
	int addr2 = (m + 1)*d - tib - 1;

	float A = smem[addr1];
	float B = smem[addr2];
	smem[addr1] = max(A, B);
	smem[addr2] = min(A, B);
}

__device__ void _bitonicStep2_fp32(float * smem, int tid, int tpp, int d)
{
	int m = tid / (d >> 1);
	int tib = tid - m*(d >> 1);
	int addr1 = d*m + tib;
	int addr2 = addr1 + (d >> 1);

	float A = smem[addr1];
	float B = smem[addr2];
	smem[addr1] = max(A, B);
	smem[addr2] = min(A, B);
}

__global__ void bitonicSortKernel128_fp32(float* mem)
{
	// Operating on 64 samples
	int bid = blockIdx.x; // Block UID
	int tpp = threadIdx.x; // Thread position in block
	int tid = blockIdx.x*blockDim.x + threadIdx.x; // Thread global UID
	__shared__ float smem[256]; // Two blocks worth of shared memory
	smem[tpp] = mem[blockDim.x*(2 * bid) + tpp]; // Coalesced memory load
	smem[tpp + blockDim.x] = mem[blockDim.x*((2 * bid) + 1) + tpp]; // Coalesced memory load
	int blocks = 8;
	for (int blockNum = 1; blockNum <= blocks; blockNum++)
	{
		int d = 1 << blockNum;
		_bitonicStep1_fp32(smem, tpp, tpp, d);
		__syncthreads();
		d = d >> 1;
		while (d >= 2)
		{
			_bitonicStep2_fp32(smem, tpp, tpp, d);
			__syncthreads();
			d = d >> 1;
		}
	}

	mem[blockDim.x*(2 * bid) + tpp] = smem[tpp];
	mem[blockDim.x*((2 * bid) + 1) + tpp] = smem[tpp + blockDim.x];
}

__global__ void bitonicSortKernelXBlock1_fp32(float* mem, int blockNum)
{
	int bid = blockIdx.x; // Block UID
	int tpp = threadIdx.x; // Thread position in block
	int tid = blockIdx.x*blockDim.x + threadIdx.x; // Thread global UID
	int d = 1 << blockNum;
	_bitonicStep1_fp32(mem, tid, tpp, d);
}
__global__ void bitonicSortKernelXBlock2_fp32(float* mem, int blockNum, int d)
{
	int bid = blockIdx.x; // Block UID
	int tpp = threadIdx.x; // Thread position in block
	int tid = blockIdx.x*blockDim.x + threadIdx.x; // Thread global UID
	_bitonicStep2_fp32(mem, tid, tpp, d);

}


// For float32

__device__ void _bitonicStep1_fp64(double * smem, int tid, int tpp, int d)
{
	int m = tid / (d >> 1);
	int tib = tid - m*(d >> 1);
	int addr1 = d*m + tib;
	int addr2 = (m + 1)*d - tib - 1;

	double A = smem[addr1];
	double B = smem[addr2];
	smem[addr1] = max(A, B);
	smem[addr2] = min(A, B);
}

__device__ void _bitonicStep2_fp64(double * smem, int tid, int tpp, int d)
{
	int m = tid / (d >> 1);
	int tib = tid - m*(d >> 1);
	int addr1 = d*m + tib;
	int addr2 = addr1 + (d >> 1);

	double A = smem[addr1];
	double B = smem[addr2];
	smem[addr1] = max(A, B);
	smem[addr2] = min(A, B);
}

__global__ void bitonicSortKernel128_fp64(double* mem)
{
	// Operating on 64 samples
	int bid = blockIdx.x; // Block UID
	int tpp = threadIdx.x; // Thread position in block
	int tid = blockIdx.x*blockDim.x + threadIdx.x; // Thread global UID
	__shared__ double smem[256]; // Two blocks worth of shared memory
	smem[tpp] = mem[blockDim.x*(2 * bid) + tpp]; // Coalesced memory load
	smem[tpp + blockDim.x] = mem[blockDim.x*((2 * bid) + 1) + tpp]; // Coalesced memory load
	int blocks = 8;
	for (int blockNum = 1; blockNum <= blocks; blockNum++)
	{
		int d = 1 << blockNum;
		_bitonicStep1_fp64(smem, tpp, tpp, d);
		__syncthreads();
		d = d >> 1;
		while (d >= 2)
		{
			_bitonicStep2_fp64(smem, tpp, tpp, d);
			__syncthreads();
			d = d >> 1;
		}
	}

	mem[blockDim.x*(2 * bid) + tpp] = smem[tpp];
	mem[blockDim.x*((2 * bid) + 1) + tpp] = smem[tpp + blockDim.x];
}

__global__ void bitonicSortKernelXBlock1_fp64(double* mem, int blockNum)
{
	int bid = blockIdx.x; // Block UID
	int tpp = threadIdx.x; // Thread position in block
	int tid = blockIdx.x*blockDim.x + threadIdx.x; // Thread global UID
	int d = 1 << blockNum;
	_bitonicStep1_fp64(mem, tid, tpp, d);
}
__global__ void bitonicSortKernelXBlock2_fp64(double* mem, int blockNum, int d)
{
	int bid = blockIdx.x; // Block UID
	int tpp = threadIdx.x; // Thread position in block
	int tid = blockIdx.x*blockDim.x + threadIdx.x; // Thread global UID
	_bitonicStep2_fp64(mem, tid, tpp, d);

}


// Helper function for using CUDA to add vectors in parallel.

hipError_t BitonicSort::BitonicSortCUDA(unsigned int* mem, int N)
{
	hipError_t cudaStatus;
	unsigned int* dev_mem;

	// Allocate GPU buffers for vector
	cudaStatus = hipMalloc((void**)&dev_mem, N * sizeof(unsigned int));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	// Copy input vectors from host memory to GPU buffers.
	cudaStatus = hipMemcpy(dev_mem, mem, N * sizeof(unsigned int), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

	// Launch a kernel on the GPU with one thread for each element.
	int numBlocks = log2(N);
	
	bitonicSortKernel128_uint32 <<<N/256, 128 >>>(dev_mem);
	for (int b = 9; b <= numBlocks; b++)
	{
		int d = 1 << b;
		bitonicSortKernelXBlock1_uint32 << <N / 512, 256 >> >(dev_mem,b);
		d = d >> 1;
		while (d >= 2)
		{
			bitonicSortKernelXBlock2_uint32 << <N / 512, 256 >> >(dev_mem, b, d);
			d = d >> 1;
		}
	}
	
	//bitonicSortKernelTestDbg <<< N / 256, 128 >>> (dev_mem);

	// Check for any errors launching the kernel
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "bitonicSortKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
		goto Error;
	}

	// hipDeviceSynchronize waits for the kernel to finish, and returns
	// any errors encountered during the launch.
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
		goto Error;
	}

	// Copy output vector from GPU buffer to host memory.
	cudaStatus = hipMemcpy(mem, dev_mem, N * sizeof(int), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}
Error:
	hipFree(dev_mem);

	return cudaStatus;
}

hipError_t BitonicSort::BitonicSortCUDA(float* mem, int N)
{
	hipError_t cudaStatus;
	float* dev_mem;

	// Allocate GPU buffers for vector
	cudaStatus = hipMalloc((void**)&dev_mem, N * sizeof(float));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	// Copy input vectors from host memory to GPU buffers.
	cudaStatus = hipMemcpy(dev_mem, mem, N * sizeof(float), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

	// Launch a kernel on the GPU with one thread for each element.
	int numBlocks = log2(N);

	bitonicSortKernel128_fp32 <<<N / 256, 128 >> >(dev_mem);
	for (int b = 9; b <= numBlocks; b++)
	{
		int d = 1 << b;
		bitonicSortKernelXBlock1_fp32 <<<N / 512, 256 >> >(dev_mem, b);
		d = d >> 1;
		while (d >= 2)
		{
			bitonicSortKernelXBlock2_fp32 <<<N / 512, 256 >> >(dev_mem, b, d);
			d = d >> 1;
		}
	}

	//bitonicSortKernelTestDbg <<< N / 256, 128 >>> (dev_mem);

	// Check for any errors launching the kernel
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "bitonicSortKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
		goto Error;
	}

	// hipDeviceSynchronize waits for the kernel to finish, and returns
	// any errors encountered during the launch.
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
		goto Error;
	}

	// Copy output vector from GPU buffer to host memory.
	cudaStatus = hipMemcpy(mem, dev_mem, N * sizeof(float), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}
Error:
	hipFree(dev_mem);

	return cudaStatus;
}

hipError_t BitonicSort::BitonicSortCUDA(double* mem, int N)
{
	hipError_t cudaStatus;
	double* dev_mem;

	// Allocate GPU buffers for vector
	cudaStatus = hipMalloc((void**)&dev_mem, N * sizeof(double));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	// Copy input vectors from host memory to GPU buffers.
	cudaStatus = hipMemcpy(dev_mem, mem, N * sizeof(double), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

	// Launch a kernel on the GPU with one thread for each element.
	int numBlocks = log2(N);

	bitonicSortKernel128_fp64 << <N / 256, 128 >> >(dev_mem);
	for (int b = 9; b <= numBlocks; b++)
	{
		int d = 1 << b;
		bitonicSortKernelXBlock1_fp64 << <N / 512, 256 >> >(dev_mem, b);
		d = d >> 1;
		while (d >= 2)
		{
			bitonicSortKernelXBlock2_fp64 << <N / 512, 256 >> >(dev_mem, b, d);
			d = d >> 1;
		}
	}

	//bitonicSortKernelTestDbg <<< N / 256, 128 >>> (dev_mem);

	// Check for any errors launching the kernel
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "bitonicSortKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
		goto Error;
	}

	// hipDeviceSynchronize waits for the kernel to finish, and returns
	// any errors encountered during the launch.
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
		goto Error;
	}

	// Copy output vector from GPU buffer to host memory.
	cudaStatus = hipMemcpy(mem, dev_mem, N * sizeof(double), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}
Error:
	hipFree(dev_mem);

	return cudaStatus;
}